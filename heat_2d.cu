#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED   0.25f

// these exist on the GPU side
texture<float,2>  texConstSrc;
texture<float,2>  texIn;
texture<float,2>  texOut;

__global__ void blend_kernel( float *dst,
                              bool dstOut ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float   t, l, c, r, b;
    if (dstOut) {
        t = tex2D(texIn,x,y-1);
        l = tex2D(texIn,x-1,y);
        c = tex2D(texIn,x,y);
        r = tex2D(texIn,x+1,y);
        b = tex2D(texIn,x,y+1);
    } else {
        t = tex2D(texOut,x,y-1);
        l = tex2D(texOut,x-1,y);
        c = tex2D(texOut,x,y);
        r = tex2D(texOut,x+1,y);
        b = tex2D(texOut,x,y+1);
    }
    dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}

__global__ void copy_const_kernel( float *iptr ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float c = tex2D(texConstSrc,x,y);
    if (c != 0)
        iptr[offset] = c;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *output_bitmap;
    float           *dev_inSrc;
    float           *dev_outSrc;
    float           *dev_constSrc;
    CPUAnimBitmap  *bitmap;

    hipEvent_t     start, stop;
    float           totalTime;
    float           frames;
};

/**
 * Draw Triangle function from Spektre's Example on stackoverflow
 * https://stackoverflow.com/a/39062479
 */
void _troj_line(int *pl,int *pr,int x0,int y0,int x1,int y1)
{
    int *pp;
    int x,y,kx,ky,dx,dy,k,m,p;
    // DDA variables (d)abs delta,(k)step direction
    kx=0; dx=x1-x0; if (dx>0) kx=+1;  if (dx<0) { kx=-1; dx=-dx; }
    ky=0; dy=y1-y0; if (dy>0) ky=+1;  if (dy<0) { ky=-1; dy=-dy; }
    // target buffer according to ky direction
    if (ky>0) pp=pl; else pp=pr;
    // integer DDA line start point
    x=x0; y=y0;
    // fix endpoints just to be sure (wrong division constants by +/-1 can cause that last point is missing)
    pp[y1]=x1; pp[y0]=x0;
    if (dx>=dy) { // x axis is major
        k=dy+dy;
        m=(dy-dx); m+=m;
        p=m;
        for (;;)
            {
            pp[y]=x;
            if (x==x1) break;
            x+=kx;
            if (p>0) { y+=ky; p+=m; } else p+=k;
            }
    }
    else {       // y axis is major
        k=dx+dx;
        m=(dx-dy); m+=m;
        p=m;
        for (;;) {
            pp[y]=x;
            if (y==y1) break;
            y+=ky;
            if (p>0) { x+=kx; p+=m; } else p+=k;
        }
    }
}

void troj(int x0, int y0, int x1, int y1, int x2, int y2, int temp, float* matrix) {
    int *pl,*pr;        // left/right buffers
    pl=new int[DIM];
    pr=new int[DIM];
    int x,y,yy0,yy1,xx0,xx1;
    // boundary line coordinates to buffers
    _troj_line(pl,pr,x0,y0,x1,y1);
    _troj_line(pl,pr,x1,y1,x2,y2);
    _troj_line(pl,pr,x2,y2,x0,y0);
    // y range
    yy0=y0; if (yy0>y1) yy0=y1; if (yy0>y2) yy0=y2;
    yy1=y0; if (yy1<y1) yy1=y1; if (yy1<y2) yy1=y2;
    // fill with horizontal lines
    for (y=yy0;y<=yy1;y++) {
        if (pl[y]<pr[y]) { xx0=pl[y]; xx1=pr[y]; }
        else             { xx1=pl[y]; xx0=pr[y]; }
        for (x=xx0;x<=xx1;x++)
            matrix[x+y*DIM]=temp;
    }
    delete[] pl;
    delete[] pr;
}

/**
 * End of Spektre's code example
 */

void triangleFan(int* xpoints, int* ypoints, int points, int temp, float* matrix) {
    for (int tri = 0; tri < points - 2; tri++) {
        troj(xpoints[0], ypoints[0],
             xpoints[tri + 1], ypoints[tri + 1],
             xpoints[tri + 2], ypoints[tri + 2],
             temp,
             matrix);
    }
}


void anim_gpu( DataBlock *d, int ticks ) {
    HANDLE_ERROR( hipEventRecord( d->start, 0 ) );
    dim3    blocks(DIM/16,DIM/16);
    dim3    threads(16,16);
    CPUAnimBitmap  *bitmap = d->bitmap;

    // since tex is global and bound, we have to use a flag to
    // select which is in/out per iteration
    volatile bool dstOut = true;
    for (int i=0; i<90; i++) {
        float   *in, *out;
        if (dstOut) {
            in  = d->dev_inSrc;
            out = d->dev_outSrc;
        } else {
            out = d->dev_inSrc;
            in  = d->dev_outSrc;
        }
        copy_const_kernel<<<blocks,threads>>>( in );
        blend_kernel<<<blocks,threads>>>( out, dstOut );
        dstOut = !dstOut;
    }
    float_to_color<<<blocks,threads>>>( d->output_bitmap,
                                        d->dev_inSrc );

    HANDLE_ERROR( hipMemcpy( bitmap->get_ptr(),
                              d->output_bitmap,
                              bitmap->image_size(),
                              hipMemcpyDeviceToHost ) );

    HANDLE_ERROR( hipEventRecord( d->stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( d->stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        d->start, d->stop ) );
    d->totalTime += elapsedTime;
    ++d->frames;
    printf( "Average Time per frame:  %3.1f ms\n",
            d->totalTime/d->frames  );
}

// clean up memory allocated on the GPU
void anim_exit( DataBlock *d ) {
    hipUnbindTexture( texIn );
    hipUnbindTexture( texOut );
    hipUnbindTexture( texConstSrc );
    HANDLE_ERROR( hipFree( d->dev_inSrc ) );
    HANDLE_ERROR( hipFree( d->dev_outSrc ) );
    HANDLE_ERROR( hipFree( d->dev_constSrc ) );

    HANDLE_ERROR( hipEventDestroy( d->start ) );
    HANDLE_ERROR( hipEventDestroy( d->stop ) );
}


int main( void ) {
    DataBlock   data;
    CPUAnimBitmap bitmap( DIM, DIM, &data );
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    HANDLE_ERROR( hipEventCreate( &data.start ) );
    HANDLE_ERROR( hipEventCreate( &data.stop ) );

    int imageSize = bitmap.image_size();

    HANDLE_ERROR( hipMalloc( (void**)&data.output_bitmap,
                               imageSize ) );

    // assume float == 4 chars in size (ie rgba)
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_inSrc,
                              imageSize ) );
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_outSrc,
                              imageSize ) );
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_constSrc,
                              imageSize ) );

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    HANDLE_ERROR( hipBindTexture2D( NULL, texConstSrc,
                                   data.dev_constSrc,
                                   desc, DIM, DIM,
                                   sizeof(float) * DIM ) );

    HANDLE_ERROR( hipBindTexture2D( NULL, texIn,
                                   data.dev_inSrc,
                                   desc, DIM, DIM,
                                   sizeof(float) * DIM ) );

    HANDLE_ERROR( hipBindTexture2D( NULL, texOut,
                                   data.dev_outSrc,
                                   desc, DIM, DIM,
                                   sizeof(float) * DIM ) );

    float *temp = (float*)malloc( imageSize );

    int points = 4;
    int xpoints[points] = {96, 48, 208, 204};
    int ypoints[points] = {740, 244, 204, 560};
    triangleFan(xpoints, ypoints, points, MAX_TEMP, temp);

    int points2 = 4;
    int xpoints2[points] = {96, 252, 548, 408};
    int ypoints2[points] = {740, 768, 208, 164};
    triangleFan(xpoints2, ypoints2, points2, MAX_TEMP, temp);

    int points3 = 4;
    int xpoints3[points] = {548, 624, 484, 452};
    int ypoints3[points] = {208, 696, 716, 388};
    triangleFan(xpoints3, ypoints3, points3, MAX_TEMP, temp);

    HANDLE_ERROR( hipMemcpy( data.dev_constSrc, temp,
                              imageSize,
                              hipMemcpyHostToDevice ) );    

    HANDLE_ERROR( hipMemcpy( data.dev_inSrc, temp,
                              imageSize,
                              hipMemcpyHostToDevice ) );
    free( temp );

    bitmap.anim_and_exit( (void (*)(void*,int))anim_gpu,
                           (void (*)(void*))anim_exit );
}

